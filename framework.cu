#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// molecule is stored as cartesian coordinates of its atom, each dimmension
// is in separate array
struct sMolecule {
	float *x;
	float *y;
	float *z;
	// some data about atoms in real application, 
	// do not corrupt mem. access optimization here...
};

#include "kernel.cu"
#include "kernel_CPU.C"

#define N 10000

void createMolecules(sMolecule A, sMolecule B, int n) {
	for (int i = 0; i < n; i++) {
		// create atom in A at random position first
		A.x[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.y[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.z[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		// create atom in B near atom A
		// in small probability, create more displaced atom
		if ((float)rand() / (float)RAND_MAX < 0.01f) {
			B.x[i] = A.x[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.y[i] = A.y[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.z[i] = A.z[i] + 10.0f * (float)rand() / (float)RAND_MAX;
		}
		else {
			B.x[i] = A.x[i] + 1.0f * (float)rand() / (float)RAND_MAX;
                        B.y[i] = A.y[i] + 1.0f * (float)rand() / (float)RAND_MAX;
                        B.z[i] = A.z[i] + 1.0f * (float)rand() / (float)RAND_MAX;
		}
	}
}

int main(int argc, char **argv){
	sMolecule A, B;
	A.x = A.y = A.z = B.x = B.y = B.z = NULL;
	sMolecule dA, dB;
	dA.x = dA.y = dA.z = dB.x = dB.y = dB.z = NULL;
	float RMSD_CPU, RMSD_GPU;

	// parse command line
	int device = 0;
	if (argc == 2) 
		device = atoi(argv[1]);
	if (hipSetDevice(device) != hipSuccess){
		fprintf(stderr, "Cannot set CUDA device!\n");
		exit(1);
	}
	hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Using device %d: \"%s\"\n", device, deviceProp.name);

	// create events for timing
	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

	// allocate and set host memory
	A.x = (float*)malloc(N*sizeof(A.x[0]));
	A.y = (float*)malloc(N*sizeof(A.y[0]));
	A.z = (float*)malloc(N*sizeof(A.z[0]));
	B.x = (float*)malloc(N*sizeof(B.x[0]));
        B.y = (float*)malloc(N*sizeof(B.y[0]));
        B.z = (float*)malloc(N*sizeof(B.z[0]));
	createMolecules(A, B, N);      
 
	// allocate and set device memory
	if (hipMalloc((void**)&dA.x, N*sizeof(dA.x[0])) != hipSuccess
	|| hipMalloc((void**)&dA.y, N*sizeof(dA.y[0])) != hipSuccess
	|| hipMalloc((void**)&dA.z, N*sizeof(dA.z[0])) != hipSuccess
	|| hipMalloc((void**)&dB.x, N*sizeof(dB.x[0])) != hipSuccess
        || hipMalloc((void**)&dB.y, N*sizeof(dB.y[0])) != hipSuccess
        || hipMalloc((void**)&dB.z, N*sizeof(dB.z[0])) != hipSuccess) {
		fprintf(stderr, "Device memory allocation error!\n");
		goto cleanup;
	}
	hipMemcpy(dA.x, A.x, N*sizeof(dA.x[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.y, A.y, N*sizeof(dA.y[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.z, A.z, N*sizeof(dA.z[0]), hipMemcpyHostToDevice);
	hipMemcpy(dB.x, B.x, N*sizeof(dB.x[0]), hipMemcpyHostToDevice);
        hipMemcpy(dB.y, B.y, N*sizeof(dB.y[0]), hipMemcpyHostToDevice);
        hipMemcpy(dB.z, B.z, N*sizeof(dB.z[0]), hipMemcpyHostToDevice);

	// solve on CPU
        printf("Solving on CPU...\n");
	hipEventRecord(start, 0);
	RMSD_CPU = solveCPU(A, B, N);
	hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);
        printf("CPU performance: %f megapairs/s\n",
                float(N)*float(N-1)/2.0f/time/1e3f);

	// solve on GPU
	printf("Solving on GPU...\n");
	hipEventRecord(start, 0);
	// run it 10x for more accurately timing results
        for (int i = 0; i < 10; i++)
		RMSD_GPU = solveGPU(dA, dB, N);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
	printf("GPU performance: %f megapairs/s\n",
                float(N)*float(N-1)/2.0f/time/1e2f);

	printf("CPU RMSD: %f\nGPU RMSD: %f\n", RMSD_CPU, RMSD_GPU);
	// check GPU results
	if ( fabsf((RMSD_CPU-RMSD_GPU) / ((RMSD_CPU+RMSD_GPU)/2)) < 0.01f)
		printf("Test OK :-).\n");
	else
		 fprintf(stderr, "Data mismatch: %f should be %f :-(\n", RMSD_GPU, RMSD_CPU);

cleanup:
	hipEventDestroy(start);
        hipEventDestroy(stop);

	if (dA.x) hipFree(dA.x);
	if (dA.y) hipFree(dA.y);
	if (dA.z) hipFree(dA.z);
	if (dB.x) hipFree(dB.x);
        if (dB.y) hipFree(dB.y);
        if (dB.z) hipFree(dB.z);
	if (A.x) free(A.x);
	if (A.y) free(A.y);
	if (A.z) free(A.z);
	if (B.x) free(B.x);
        if (B.y) free(B.y);
        if (B.z) free(B.z);

	return 0;
}

