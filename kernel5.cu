#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


#define pow_2(x) ( ((x) * (x)) )
#define BLOCK_SIZE 128

struct Atom {
    float x, y, z;
    Atom() {}
     __device__
    Atom(float x_, float y_, float z_) 
     : x(x_), y(y_), z(z_) {}
};

__global__
void atoms_difference(sMolecule A, sMolecule B,
                        float * d_result,
                        int n,
                        int line_blocks)
{
    float a_x, a_y, a_z, b_x, b_y, b_z;
    __shared__ int skip, quot, reminder;
    if (0 == threadIdx.x) {
        quot     = blockIdx.x / line_blocks;
        reminder = blockIdx.x % line_blocks;
        if (quot > reminder) {
            skip = 1;
        } else {
            skip = 0;
        }
    }
    __syncthreads();
    if (skip == 1) {
        return;
    }

    int i     = (quot) * BLOCK_SIZE + threadIdx.x;
    int begin = (reminder) * BLOCK_SIZE;

    __shared__ float A_x[BLOCK_SIZE], A_y[BLOCK_SIZE], A_z[BLOCK_SIZE];
    A_x[threadIdx.x] = A.x[begin + threadIdx.x];
    A_y[threadIdx.x] = A.y[begin + threadIdx.x];
    A_z[threadIdx.x] = A.z[begin + threadIdx.x];

    __shared__ float B_x[BLOCK_SIZE], B_y[BLOCK_SIZE], B_z[BLOCK_SIZE];
    B_x[threadIdx.x] = B.x[begin + threadIdx.x];
    B_y[threadIdx.x] = B.y[begin + threadIdx.x];
    B_z[threadIdx.x] = B.z[begin + threadIdx.x];

    __syncthreads();

    if (i >= n) {
        return;
    }

    a_x = A.x[i];
    a_y = A.y[i];
    a_z = A.z[i];

    b_x = B.x[i];
    b_y = B.y[i];
    b_z = B.z[i];

    float sum = 0.0;
    for (int j = 0; j < BLOCK_SIZE; ++j) {
        int index = begin + j;
        if (index >= n) {
            break;
        }
        if (i < index) { 
            // printf("processing (%d, %d)\n", i, index);
            float da = sqrt(pow_2(A_x[j] - a_x)
                + pow_2(A_y[j] - a_y)
                + pow_2(A_z[j] - a_z));
            float db = sqrt(pow_2(B_x[j] - b_x)
                + pow_2(B_y[j] - b_y)
                + pow_2(B_z[j] - b_z));
            // printf("Ax diff [%f, %f, %f]\n",
            //             pow_2(A.x[i] - A.x[j]),
            //             pow_2(A.y[i] - A.y[j]),
            //             pow_2(A.z[i] - A.z[j]));
            // printf("Da: %f db: %f\n", da, db);
            // printf("saving result: %f\n", pow_2(da - db));
            sum += pow_2(da - db);
        }
    }
    atomicAdd(d_result + i, sum);
}


float solveGPU(sMolecule d_A, sMolecule d_B, int n) {

    int line_blocks = n / BLOCK_SIZE + 1;
    int GRID_SIZE   = pow_2(line_blocks);

    float *d_result;
    int result_size = n;

    hipError_t err = hipMalloc(&d_result, result_size * sizeof(float));
    if ( hipSuccess != err ) {
        fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
                 __FILE__, __LINE__, hipGetErrorString(err) );
        return 0.0f;
    }
    err = hipMemset(d_result, 0, result_size * sizeof(float));
    if ( hipSuccess != err ) {
        fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
                 __FILE__, __LINE__, hipGetErrorString(err) );
        return 0.0f;
    }

    atoms_difference<<<GRID_SIZE, BLOCK_SIZE>>>
                    (d_A, d_B, d_result, n, line_blocks);

    float RMSD = 0;
    thrust::device_ptr<float> dptr(d_result);
    RMSD = thrust::reduce(thrust::device, dptr, dptr + result_size);

    hipFree(d_result);
    return sqrt(1 / ((float)n * ((float)n - 1)) * RMSD);
}
