#include "hip/hip_runtime.h"
#include <hipblas.h>

#define pow_2(x) ( ((x) * (x)) )

__global__
void atoms_difference(sMolecule A, sMolecule B,
                        float * d_result,
                        int width,
                        int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i * width + j < size) {
        // printf("processing (%d, %d)\n", i, j);
        if (i < width && i < j && j < width) {
            float da = sqrt(pow_2(A.x[i] - A.x[j])
                    + pow_2(A.y[i] - A.y[j])
                    + pow_2(A.z[i] - A.z[j]));
            float db = sqrt(pow_2(B.x[i] - B.x[j])
                + pow_2(B.y[i] - B.y[j])
                + pow_2(B.z[i] - B.z[j]));
            // printf("Ax diff [%f, %f, %f]\n",
            //             pow_2(A.x[i] - A.x[j]),
            //             pow_2(A.y[i] - A.y[j]),
            //             pow_2(A.z[i] - A.z[j]));
            // printf("Da: %f db: %f\n", da, db);
            // printf("saving result: %f\n", pow_2(da - db));
            d_result[i * width + j] = pow_2(da - db);
        } else {
            d_result[i * width + j] = 0.0f;
        }
    }
}

float * array_malloc_GPU(int size)
{
    float * d_array;
    hipMalloc(&d_array, size * sizeof(float));
    return d_array;
}

float * array_to_GPU(float * array, int size)
{
    float * d_array = array_malloc_GPU(size);
    hipMemcpy(d_array, array,
                size * sizeof(float),
                hipMemcpyHostToDevice);
    return d_array;
}

sMolecule molecule_to_GPU(sMolecule A, int size)
{
    sMolecule d_A;
    d_A.x = array_to_GPU(A.x, size);
    d_A.y = array_to_GPU(A.y, size);
    d_A.z = array_to_GPU(A.z, size);
    return d_A;
}

void free_molecule(sMolecule d_A)
{
    hipFree(d_A.x);
    hipFree(d_A.y);
    hipFree(d_A.z);
}

float solveGPU(sMolecule d_A, sMolecule d_B, int n) {

    int BLOCK_SIZE  = 8;
    int result_size = pow_2(n);

    float *d_result;

    hipError_t err = hipMalloc(&d_result, result_size * sizeof(float));
    if ( hipSuccess != err ) {
        fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
                 __FILE__, __LINE__, hipGetErrorString( err) );
        return 0.0f;
    }

    int GRID_SIZE = (n / BLOCK_SIZE) + 1;
    printf("Grid size: %d\n", GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);

    atoms_difference<<<dimGrid, dimBlock>>>
                    (d_A, d_B, d_result, n, result_size);;

    hipblasStatus_t ret;
    hipblasHandle_t handle;
    ret = hipblasCreate(&handle);

    float RMSD = 0.0f;
    // sum using cublas reduction algorithm
    hipblasSasum(handle, result_size, d_result, 1, &RMSD);

    hipFree(d_result);
	return sqrt(1 / ((float)n * ((float)n - 1)) * RMSD);
}
